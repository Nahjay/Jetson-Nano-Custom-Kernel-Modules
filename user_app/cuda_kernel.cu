#include "hip/hip_runtime.h"
#include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
#include "cuda_kernel.h"
#include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
#include "/usr/local/cuda-10.2/include/"

#define IMAGE_WIDTH 1280
#define IMAGE_HEIGHT 853
#define METADATA 100
// Updated CUDA kernel function to process image data
__global__ void cuda_kernel(char *image_data, size_t width, size_t height, size_t metadata_size) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Skip modifying metadata
    if (index < metadata_size) {
        return;
    }

    // Calculate pixel index
    int pixel_index = (index - metadata_size) / 3;

    // Calculate channel index
    int channel_index = (index - metadata_size) % 3;

    // Ensure index is within image data bounds
    if (pixel_index < width * height) {
        // Increment RGB values by 10
        image_data[index] = (image_data[index] + 10) % 256;
    }
}

// Function to process image data using Cuda
extern "C" void process_image_data(char *image_data) {


    // Get the size of the image data
    size_t image_data_size = sizeof(*image_data);

    // Create pointers to the image data
    char *d_image_data;

    // Allocate memory for the image data on the device
    hipMalloc((void**) &d_image_data, image_data_size);

    // Check if memory was allocated successfully
    if (d_image_data == NULL) {
        fprintf(stderr, "Failed to allocate memory for image data on device\n");
        exit(EXIT_FAILURE);

        // cleanup
        hipFree(d_image_data);
        d_image_data = NULL;
    }
    else {
        printf("Successfully allocated memory for image data on device\n");
    }

    // Copy image data from host to device
    hipMemcpy(d_image_data, image_data, image_data_size, hipMemcpyHostToDevice);

    // Check if image data was copied successfully
    if (hipMemcpy(d_image_data, image_data, image_data_size, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Failed to copy image data from host to device\n");
        exit(EXIT_FAILURE);

        // cleanup
        hipFree(d_image_data);
        d_image_data = NULL;
    }
    else {
        printf("Successfully copied image data from host to device\n");
    }

    // Define grid and block dimensions
    int blockSize = 256;
    int gridSize = (image_data_size + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    cuda_kernel<<<gridSize, blockSize>>>(d_image_data, IMAGE_WIDTH, IMAGE_HEIGHT, METADATA);
    // Check for kernel launch errors
    hipError_t cuda_error = hipGetLastError();
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch failed: %s\n", hipGetErrorString(cuda_error));
        exit(EXIT_FAILURE);
    }

    // Copy the modified image data back to host memory
    hipMemcpy(image_data, d_image_data, image_data_size, hipMemcpyDeviceToHost);

    // Check if image data was copied back successfully
    cuda_error = hipGetLastError();
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "Failed to copy modified image data back to host: %s\n", hipGetErrorString(cuda_error));
        exit(EXIT_FAILURE);
    }

    // Free device memory
    hipFree(d_image_data);
}

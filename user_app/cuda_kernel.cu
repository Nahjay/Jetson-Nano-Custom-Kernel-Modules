#include "hip/hip_runtime.h"
// #include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
// #include "cuda_kernel.h"
// #include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
// #include "/usr/local/cuda-10.2/include/"
// #include <stdio.h>
// #include <stdlib.h>


// #define IMAGE_WIDTH 1280
// #define IMAGE_HEIGHT 853
// #define METADATA 100
// // Updated CUDA kernel function to process image data
// __global__ void cuda_kernel(char *image_data, size_t width, size_t height, size_t metadata_size) {
//     // Calculate global index
//     int index = blockIdx.x * blockDim.x + threadIdx.x;

//     // Skip modifying metadata
//     if (index < metadata_size) {
//         return;
//     }

//     // Calculate pixel index
//     int pixel_index = (index - metadata_size) / 3;

//     // Ensure index is within image data bounds
//     if (pixel_index < width * height) {
//         // Increment RGB values by 10
//         image_data[index] = (image_data[index] + 10) % 256;
//     }
// }

// // Function to process image data using Cuda
// void process_image_data(char *image_data) {


//     // Get the size of the image data
//     size_t image_data_size = sizeof(*image_data);

//     // Create pointers to the image data
//     char *d_image_data;

//     // Allocate memory for the image data on the device
//     hipMalloc((void**) &d_image_data, image_data_size);

//     // Check if memory was allocated successfully
//     if (d_image_data == NULL) {
//         fprintf(stderr, "Failed to allocate memory for image data on device\n");
//         exit(EXIT_FAILURE);

//         // cleanup
//         hipFree(d_image_data);
//         d_image_data = NULL;
//     }
//     else {
//         printf("Successfully allocated memory for image data on device\n");
//     }

//     // Copy image data from host to device
//     hipMemcpy(d_image_data, image_data, image_data_size, hipMemcpyHostToDevice);

//     // Check if image data was copied successfully
//     if (hipMemcpy(d_image_data, image_data, image_data_size, hipMemcpyHostToDevice) != hipSuccess) {
//         fprintf(stderr, "Failed to copy image data from host to device\n");
//         exit(EXIT_FAILURE);

//         // cleanup
//         hipFree(d_image_data);
//         d_image_data = NULL;
//     }
//     else {
//         printf("Successfully copied image data from host to device\n");
//     }

//     // Define grid and block dimensions
//     int blockSize = 256;
//     int gridSize = (image_data_size + blockSize - 1) / blockSize;

//     // Launch the CUDA kernel
//     cuda_kernel<<<gridSize, blockSize>>>(d_image_data, IMAGE_WIDTH, IMAGE_HEIGHT, METADATA);
//     // Check for kernel launch errors
//     hipError_t cuda_error = hipGetLastError();
//     if (cuda_error != hipSuccess) {
//         fprintf(stderr, "CUDA kernel launch failed: %s\n", hipGetErrorString(cuda_error));
//         exit(EXIT_FAILURE);
//     }

//     // Copy the modified image data back to host memory
//     hipMemcpy(image_data, d_image_data, image_data_size, hipMemcpyDeviceToHost);

//     // // Write the modified image data to a file
//     // FILE *output_file = fopen("output_image.ppm", "wb");
//     // fwrite(image_data, sizeof(char), image_data_size, output_file);
//     // fclose(output_file);

//     // Write the modified image data to a file
//     FILE *output_file = fopen("output_image.ppm", "wb");
//     if (output_file == NULL) {
//         fprintf(stderr, "Failed to open output file\n");
//         exit(EXIT_FAILURE);
//     }

//     // Write the PPM header
//     fprintf(output_file, "P6\n%d %d\n255\n", IMAGE_WIDTH, IMAGE_HEIGHT);

//     // Write the image data to the file
//     size_t pixel_count = IMAGE_WIDTH * IMAGE_HEIGHT * 3; // Assuming 3 bytes per pixel (RGB)
//     size_t bytes_written = fwrite(image_data, sizeof(char), pixel_count, output_file);
//     if (bytes_written != pixel_count) {
//         fprintf(stderr, "Error writing image data to file\n");
//         exit(EXIT_FAILURE);
//     }

//     // Close the file
//     fclose(output_file);

//     // Check if image data was copied back successfully
//     cuda_error = hipGetLastError();
//     if (cuda_error != hipSuccess) {
//         fprintf(stderr, "Failed to copy modified image data back to host: %s\n", hipGetErrorString(cuda_error));
//         exit(EXIT_FAILURE);
//     }

//     // Free device memory
//     hipFree(d_image_data);

//     // Check if device memory was freed successfully
//     cuda_error = hipGetLastError();

//     if (cuda_error != hipSuccess) {
//         fprintf(stderr, "Failed to free device memory: %s\n", hipGetErrorString(cuda_error));
//         exit(EXIT_FAILURE);
//     }
//     else {
//         printf("Successfully freed device memory\n");
//     }

//     // Sync the device
//     hipDeviceSynchronize();
 
// }


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
#include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
#include "/usr/local/cuda-10.2/include/"

#define MAX_PATH 256
#define METADATA 100
#define IMAGE_WIDTH 1280
#define IMAGE_HEIGHT 853
#define IMAGE_SIZE (IMAGE_WIDTH * IMAGE_HEIGHT * 3) // Assuming RGB format
#define BLOCK_SIZE 256

__global__ void modify_ppm_colors(unsigned char *data, size_t width, size_t height, size_t metadata_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    while (idx < width * height) {
        int y = idx / width;
        int x = idx % width;
        size_t index = y * width * 3 + x * 3;

        // Skip modifying metadata
        if (index >= metadata_size) {
            // Increment RGB values by 10
            for (int channel = 0; channel < 3; ++channel) {
                data[index + channel] = (data[index + channel] + 10) % 256;
            }
        }

        idx += blockDim.x * gridDim.x;
    }
}

int process_image_data(char* argv[]) {
    // Open the file
    FILE *file = fopen(argv[2], "rb+");
    if (!file) {
        perror("Error opening file");
        return 1;
    }

    unsigned char *data = (unsigned char *)malloc(IMAGE_SIZE * sizeof(unsigned char));
    if (!data) {
        perror("Error allocating memory");
        fclose(file);
        return 1;
    }

    // Read the data from the file
    size_t bytesRead = fread(data, sizeof(unsigned char), IMAGE_SIZE, file);
    if (bytesRead != IMAGE_SIZE) {
        perror("Error reading file");
        fclose(file);
        free(data);
        return 1;
    }

    // Close the file
    fclose(file);

    // Launch CUDA kernel
    unsigned char *d_data;
    hipMalloc(&d_data, IMAGE_SIZE * sizeof(unsigned char));
    hipMemcpy(d_data, data, IMAGE_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice);
    modify_ppm_colors<<<(IMAGE_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_data, IMAGE_WIDTH, IMAGE_HEIGHT, METADATA);
    hipMemcpy(data, d_data, IMAGE_SIZE * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipFree(d_data);

    // Write back modified data to the file
    FILE *output_file = fopen(argv[2], "wb");
    if (!output_file) {
        perror("Error opening output file");
        free(data);
        return 1;
    }
    fwrite(data, sizeof(unsigned char), IMAGE_SIZE, output_file);
    fclose(output_file);

    // Free allocated memory
    free(data);

    return 0;
}